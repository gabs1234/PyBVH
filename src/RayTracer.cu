#include "hip/hip_runtime.h"
#include "RayTracer.cuh"

// #include <thrust/sort.h>

__host__ __device__ RayTracer::RayTracer(BVHTree *tree, float4 *vertices, unsigned int nb_vertices) {
    this->tree = tree;
    this->vertices = vertices;
    this->nbVertices = nb_vertices;
    this->raySet = 0;
}

__host__ __device__ RayTracer::RayTracer(BVHTree *tree, float4 origin, float4 *vertices, unsigned int nb_vertices, bool parallel) {
    this->tree = tree;
    this->origin = origin;
    this->vertices = vertices;
    this->nbVertices = nb_vertices;
    this->raySet = 0;
    this->parallelGeometry = parallel;
}

__host__ __device__ float4 RayTracer::sphericalToCartesian(float theta, float phi, float r) {
    float const x = r * sin(theta) * cos(phi);
    float const y = r * sin(theta) * sin(phi);
    float const z = r * cos(theta);

    return make_float4(x, y, z, 1.0);
}



__host__ __device__ bool RayTracer::computeRayAABB(float4 &O, float4 &min, float4 &max) {
    float4 sceneBBMin = this->tree->getSceneBBMin();
    float4 sceneBBMax = this->tree->getSceneBBMax();
    float4 rayBBmin, rayBBmax;
    float2 t;
    
    if (this->ray.intersects(sceneBBMin, sceneBBMax, t)) {
        min = this->ray.computeParametric(t.x);
        max = this->ray.computeParametric(t.y);
        return true;
    }
    return false;
}

__host__ __device__ BasisNamespace::Basis RayTracer::makeProjectionBasis (BasisNamespace::Basis &MeshBasis, float4 &spherical, float4 &euler) {
    // Precompute cos and sin values
    float cos_theta = cos(spherical.x);
    float cos_phi = cos(spherical.y);
    float sin_theta = sin(spherical.x);
    float sin_phi = sin(spherical.y);

    // Compute the BasisNamespace::Basisbasis vectors
    float4 w = make_float4(cos_phi * sin_theta, sin_phi * sin_theta, cos_theta, 0.0);
    float4 u = make_float4(cos_phi * cos_theta, sin_phi * cos_theta, -sin_theta, 0.0);
    float4 v = make_float4(-sin_phi, cos_phi, 0.0, 0.0);

    // Get origin of the BasisNamespace::Basisbasis
    float4 meshOrigin = MeshBasis.getOrigin();
    float4 new_origin;

    new_origin.x = meshOrigin.x + spherical.z * w.x;
    new_origin.y = meshOrigin.y + spherical.z * w.y;
    new_origin.z = meshOrigin.z + spherical.z * w.z;
    new_origin.w = 0;

    // w = -w
    w.x = -w.x;
    w.y = -w.y;
    w.z = -w.z;
    u.x = -u.x;
    u.y = -u.y;
    u.z = -u.z;
    v.x = -v.x;
    v.y = -v.y;
    v.z = -v.z;

    // Create the new BasisNamespace::Basisbasis
    BasisNamespace::Basis new_basis = BasisNamespace::Basis(new_origin, u, v, w);

    // Rotate the BasisNamespace::Basisbasis
    new_basis.rotate(euler);

    return new_basis;
}

__host__ __device__ float computeThickness(CollisionList &tvalues) {
    

    int i, j;
    float result = 0.0;

    i = 0;
    while (i < tvalues.count) {
        j = i + 1;
        while (j < tvalues.count && fabsf (tvalues.collisions[j] - tvalues.collisions[i]) < 0.0000001) {
            j++;
        }
        if (i < tvalues.count && j < tvalues.count) {
            result += fabsf (tvalues.collisions[j] - tvalues.collisions[i]);
        }
        i = j + 1;
    }

    return result;
}

__host__ __device__ float sumTvalues (CollisionList &t_values) {
    float thickness = 0;
    for (int i = 0; i < t_values.count; i++) {
        thickness += t_values.collisions[i];
    }
    return thickness;
}

__host__ __device__ float RayTracer::traceRayParallel(Ray &ray) {
    

    CollisionList candidates;
    candidates.count = 0;
    memset(candidates.collisions, 0, MAX_COLLISIONS * sizeof(float));

    CollisionList tvalues;
    tvalues.count = 0;
    memset(tvalues.collisions, 0, MAX_COLLISIONS * sizeof(float));

    // This is where the acceleration structure (BVH) is actually usefull
    this->tree->query(ray, candidates);

    if (candidates.count == 0) {
        return 0.0;
    }
    else {
        // printf ("candidates count = %d\n", candidates.count);
    }

    // Test the candidates for actual intersections
    for (int i = 0; i < candidates.count; i++) {
        int primIndex = candidates.collisions[i]*3;

        if (primIndex + 2>= this->nbVertices || primIndex < 0) {
            continue;
        }
        
        // printf("Collision at %d\n", primIndex);
        float4 V1 = this->vertices[primIndex];
        float4 V2 = this->vertices[primIndex + 1];
        float4 V3 = this->vertices[primIndex + 2];

        float t;
        if (ray.intersects(V1, V2, V3, t)) {
            // printf("real Collision at %d, %f\n", primIndex, t);
            tvalues.collisions[tvalues.count++] = t;
        }
    }

    // Print the t_values


    // Sort the t_values
    // thrust::sort(thrust::device, candidates.collisions, candidates.collisions + candidates.count);

    // printf ("t_values count = %d\n", t_values.count);
    // for (int i = 0; i < t_values.count; i++) {
    //     printf ("t_values[%d] = %f\n", i, t_values.collisions[i]);
    // }

    // compute the thickness
    return sumTvalues(tvalues);
}

__host__ __device__ void RayTracer::testSingleRay(Ray ray, CollisionList *collisions) {
    this->tree->query(ray, *collisions);
}

__global__ void projectPlaneRaysKernel (
    RayTracer *tracer, float *image, 
    uint2 N, float2 D,
    float4 spherical, float4 euler, float4 meshOrigin) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (N.x == 0 || N.y == 0) {
        return;
    }

    if (N.x * N.y <= tid) {
        return;
    }

    BasisNamespace::Basis meshBasis = BasisNamespace::Basis(
        meshOrigin,
        make_float4(1, 0, 0, 0),
        make_float4(0, 1, 0, 0),
        make_float4(0, 0, 1, 0));

    BasisNamespace::Basis projectionPlaneBasis = tracer->makeProjectionBasis(
        meshBasis, spherical, euler);
        
    float delta_x = D.x / (N.x-1);
    float delta_y = D.y / (N.y-1);
    projectionPlaneBasis.scale(delta_x , delta_y , 1);

    if (tid == 0) {
        meshBasis.print();
        projectionPlaneBasis.print();
    }

    for (size_t gid = tid; gid < N.x * N.y; gid += blockDim.x * gridDim.x) {
        int i = gid % N.x;
        int j = gid / N.x;
        // printf ("i = %d, j = %d\n", i, j);
        float4 ray_origin = projectionPlaneBasis.getPointInBasis(make_float4(i, j, 0, 0));
        
        Ray ray = Ray(ray_origin, projectionPlaneBasis.getVector(2)); // ray along z axis

        // ray.print();

        // printf ("ray origin = (%f, %f, %f)\n", ray_origin.x, ray_origin.y, ray_origin.z);

        image[gid] = tracer->traceRayParallel(ray);
    }
}