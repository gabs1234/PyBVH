#include "hip/hip_runtime.h"
#include <hipcub/hipcub.hpp>
#include <hip/hip_runtime_api.h>

//---------------------------------------------------------------------
// Kernels
//---------------------------------------------------------------------

#include <>
#include <hip/hip_runtime.h>
/**
 * Simple kernel for performing a block-wide sorting over integers
 */



// In and out buffers may be swaped
// Original data is not kept
extern "C" {

#define KEY_TYPE unsigned int
#define VALUE_TYPE unsigned int

__global__ void deviceSort(
    unsigned int numberOfElements, 
    KEY_TYPE** keysIn, KEY_TYPE** keysOut,
    VALUE_TYPE** valuesIn, VALUE_TYPE** valuesOut)
{
    hipcub::DoubleBuffer<KEY_TYPE> keysBuffer(*keysIn, *keysOut);
    hipcub::DoubleBuffer<VALUE_TYPE> valuesBuffer(*valuesIn, *valuesOut);

    // Check how much temporary memory will be required
    void* tempStorage = nullptr;
    size_t storageSize = 0;
    // hipcub::DeviceRadixSort::SortPairs(tempStorage, storageSize, keysBuffer, valuesBuffer,
    // numberOfElements);
    hipcub::DeviceRadixSort::SortKeys(tempStorage, storageSize, keysBuffer, numberOfElements);

    // Allocate temporary memory
    hipMalloc(&tempStorage, storageSize);

    // Sort
    hipcub::DeviceRadixSort::SortPairs(tempStorage, storageSize, keysBuffer, valuesBuffer,
                                    numberOfElements);

    // Free temporary memory
    hipFree(tempStorage);

    // Update out buffers
    KEY_TYPE* current = keysBuffer.Current();
    keysOut = &current;
    unsigned int* current2 = valuesBuffer.Current();
    valuesOut = &current2;

    // Update in buffers
    current = keysBuffer.d_buffers[1 - keysBuffer.selector];
    keysIn = &current;
    current2 = valuesBuffer.d_buffers[1 - valuesBuffer.selector];
    valuesIn = &current2;
}

}